#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

#define N 1000	//NUMERO MAXIMO DE BLOQUES
#define T 1		//NUMERO MAXIMO DE HILOS POR BLOQUE
#define V 1000	//TAMAÑO DE LOS VECTORES

__global__ void vecAdd(int *a, int *b, int *c);

int main() {
	int a[V], b[V], c[V];
	int *dev_a, *dev_b, *dev_c;

	for (int i = 0; i<V; i++) {
		a[i] = i;
		b[i] = i; 
	}
	
	hipMalloc((void**)&dev_a, V * sizeof(int));
	hipMalloc((void**)&dev_b, V * sizeof(int));
	hipMalloc((void**)&dev_c, V * sizeof(int));
	
	hipMemcpy(dev_a, a, V * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, V * sizeof(int), hipMemcpyHostToDevice);
	
	dim3 bDim(N,1);
	dim3 bGrid(T,1);
	
	vecAdd << < N , T >> >(dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, V * sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	printf("El resultado de las operaciones :\n");
	for (int i = 0; i<V; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	return 0;
}
__global__ void vecAdd(int *a, int *b, int *c) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while(tid < N)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}