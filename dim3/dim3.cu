#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include <stdio.h>

//#define N 65535 //Bloques
//#define T 128 //Hilos
//#define D N*T
#define D 8388480

int a[D], b[D], c[D];

__global__ void vecAdd(int *a, int *b, int *c);

int main() {
	int *dev_a, *dev_b, *dev_c;

	for (int i = 0; i<D; i++) {
		a[i] = i;
		b[i] = i; 

	}

	hipMalloc((void**)&dev_a, D * sizeof(int));
	hipMalloc((void**)&dev_b, D * sizeof(int));
	hipMalloc((void**)&dev_c, D * sizeof(int));

	hipMemcpy(dev_a, a, D * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, D * sizeof(int), hipMemcpyHostToDevice);
	
	dim3 bDim(65535,1);
	dim3 bGrid(128,1);

	vecAdd << <bDim , bGrid >> >(dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, D * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	printf("El resultado de las operaciones :\n");
	for (int i = 0; i<D; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	return 0;
}
__global__ void vecAdd(int *a, int *b, int *c) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while(tid < D)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}

}
