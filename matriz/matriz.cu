#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>     /* malloc, free, rand */

#define D 256

float entrada[D], salida[D];
__global__ void transponer(float* entrada, float* salida, int ANCHO){
	int tx = blockIdx.x*blockDim.x + threadIdx.x;
	int ty = blockIdx.y*blockDim.y + threadIdx.y;
	salida[tx*ANCHO + ty] = entrada[ty*ANCHO + tx];
}
int main(int args, char* argv[]){

	for (int i = 0; i<D; i++) 
	{
		entrada[i] = i;	
	}
	const int ALTO = 16;
	const int ANCHO = 16;
	const int TAM = ALTO*ANCHO*sizeof(float);

	float* M = (float*)malloc(TAM);
	float* Md = NULL;
	float* ld = NULL;
	hipMalloc((void**)&Md, TAM);
	hipMalloc((void**)&ld, TAM);

	hipMemcpy(Md, entrada, D * sizeof(float), hipMemcpyHostToDevice);
	
	dim3 bDim(16, 16);
	dim3 gDim(ALTO / bDim.x, ALTO / bDim.y);
	transponer << < gDim, bDim >> > (Md, ld, ANCHO);
	hipMemcpy(salida, ld, D * sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(Md);
	hipFree(ld);

	printf("El resultado de las operaciones :\n");
	for (int i = 0; i<D; i++)
	{
		printf(" entrada = %f | salida = %f \n",entrada[i],salida[i]);
	}

	
	
	free(M);
	return 0;
}