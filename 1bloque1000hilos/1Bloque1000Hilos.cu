#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#define N 1 //BLOQUES
#define T 1000 //NUMERO MAXIMO DE HILOS POR BLOQUE

__global__ void vecAdd(int *a, int *b, int *c);

int main() {
	int a[T], b[T], c[T];
	int *dev_a, *dev_b, *dev_c;
	for (int i = 0; i<T; i++) {
		a[i] = i;
		b[i] = i*i; 
    }
    
	hipMalloc((void**)&dev_a, T * sizeof(int));
	hipMalloc((void**)&dev_b, T * sizeof(int));
    hipMalloc((void**)&dev_c, T * sizeof(int));
    
	hipMemcpy(dev_a, a, T * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, T * sizeof(int), hipMemcpyHostToDevice);

	vecAdd << <N , T >> >(dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, T * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
    hipFree(dev_c);
    
	printf("El resultado de las operaciones :\n");
	for (int i = 0; i<T; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	return 0;
}
__global__ void vecAdd(int *a, int *b, int *c) {
	int i = blockIdx.x;
	if (i < T) 
	{
		c[i] = a[i] + b[i];
	}
}