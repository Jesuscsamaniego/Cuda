#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

int main(void) 
{
	hipDeviceProp_t  prop;
	int count;
	hipGetDeviceCount(&count);
	
	for (int i = 0; i< count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("   --- Informacion general de los dispositivos %d ---\n", i);
		printf("Nombre:  %s\n", prop.name);
		printf("Capacidad de computo:  %d.%d\n", prop.major, prop.minor);
		printf("Velocidad del Reloj:  %d mhz\n", prop.clockRate);
		
		printf("Sobre posicion de copia del dispositivo:  ");
		if (prop.deviceOverlap){
			printf("Habilidada\n");
			printf("Ejecucion simultanea de hipMemcpy() y un kernel\n");
		}
		else
		{
			printf("Deshabilitada\n");
		}
		
		printf("Limite de tiempo de ejecucion del kernel :  ");
		if (prop.kernelExecTimeoutEnabled){
			printf("Habilitada\n");
		}
		else{
			printf("Deshabilidad\n");
		}

		printf("Soporte de kernels concurrentes :  ");
		if (prop.concurrentKernels){
			printf("Habilitada\n");
		}
		else{
			printf("Deshabilidad\n");
		}

		printf("Memoria con codigo de correccion de errores :  ");
		if (prop.ECCEnabled){
			printf("Habilitada\n");
		}
		else{
			printf("Deshabilidad\n");
		}
		
		printf("   --- Informacion de memoria del dispositivo %d ---\n", i);
		printf("Memoria global total:  %ld bytes\n", prop.totalGlobalMem);
		printf("Memoria total para constantes:  %ld\n", prop.totalConstMem);
		printf("Pico max en copias de memoria (bytes):  %ld\n", prop.memPitch);
		printf("Alineamiento de texturas:  %ld\n", prop.textureAlignment);
		printf("   --- Informacion secundaria del dispositivo %d ---\n", i);
		printf("Conteo de multiprocesadores en el GPU:  %d\n",
			prop.multiProcessorCount);
		printf("Memoria compartida por multiprocesadores:  %ld\n", prop.sharedMemPerBlock);
		printf("Registros por  mp:  %d\n", prop.regsPerBlock);
		printf("hilos por grupo:  %d\n", prop.warpSize);
		printf("Cantidad maxima de hilos por bloque:  %d\n",
			prop.maxThreadsPerBlock);
		printf("Dimensiones maximas del hilo:  (%d, %d, %d)\n",
			prop.maxThreadsDim[0], prop.maxThreadsDim[1],
			prop.maxThreadsDim[2]);
		printf("Tamanio maximo de los GRIDS:  (%d, %d, %d)\n",
			prop.maxGridSize[0], prop.maxGridSize[1],
			prop.maxGridSize[2]);
		printf("\n");
	}
	
}