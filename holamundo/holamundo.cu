#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
__global__ void mikernel(void){
	printf("Llamada a kernel de gpu\n");
}
main(void){
	mikernel << <1, 1 >> >();
	hipDeviceSynchronize();
	printf("Mensaje desde el cpu \n");
	return 0;
}
